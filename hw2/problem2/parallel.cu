#include "hip/hip_runtime.h"
/* Daniel Willen, 2019
 *
 * Solve the transient heat conduction problem with homogeneous Dirichlet
 *  boundary conditions:
 *
 *    u(x={0,L}) = u(y={0,L}) = 0
 *
 *  and initial condition:
 *
 *    u(x,y,0) = sin(x) * sin(y)
 *
 *  on the domain 0 <= x,y <= L, with L = pi.
 *
 * This program solves the above problem on a single GPU with the Jacobi method.
 * 
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#define PI 3.14159265358979323846
#define MAX_THREADS_DIM 16        // Note that this depends on the hardware

/* Note on the structure of this file:
 *  - Cuda device constant memory declarations are at the top
 *  - Functions definitions are in the middle. Functions include:
 *  - - parse_cmdline: Read command-line arguments for domain size
 *  - - jacobi_solver: Advance the soln to the next time step using Jacobi
 *  - - check_error:   Calculate the error b/t the numeric and analytic solns
 *  - The `main' function is at the bottom
 *
 *  Note that it is good practice to use header files and break functions out
 *   into separate files. This has not been done here for simplicity.
 */

/*** Auxiliary Functions ***/

/* Read the command line inputs */
// - argv[0] is the program name
// - argv[1] is the first input (number of points)
int parse_cmdline(int argc, char *argv[]) {
  int nx;
  if (argc == 2) {
    nx = atoi(argv[1]); // Number of grid points
		if (nx < MAX_THREADS_DIM) {
			printf("Expecting a number of grid cells in one dimension to be at least %d\n", MAX_THREADS_DIM);
			exit(EXIT_FAILURE);
		}

    printf("Grid is %d by %d\n\n", nx, nx);
  } else {
    printf("Input error. Run like: \n\n");
    printf("  $ ./parallel.c n\n\n");
    printf("  where n is the number of grid cells in one dimension\n");
    exit(EXIT_FAILURE);
  }
  return nx;
}

/*******************************************************************************
 * Step IV: Launch the GPU kernel to advance to the next time step with the    *
 *          Jacobi method here.                                                *
 ******************************************************************************/
__global__ void computeNextJacobiStep(int nx, int ny, double pref, double* _u, double* _u_new) {
	int ti = blockDim.x * blockIdx.x + threadIdx.x;
	int tj = blockDim.y * blockIdx.y + threadIdx.y;

	if (ti < (nx-1) && ti > 0 && tj < (ny-1) && tj > 0) {
		double leftTerm = _u[tj*nx + ti];
		double rightTerm = pref * (
			_u[tj*nx + (ti+1)] +
			_u[tj*nx + (ti-1)] +
			_u[(tj+1)*nx + ti] +
			_u[(tj-1)*nx + ti] -
			4*_u[tj*nx + ti]
		);
		
		_u_new[tj*nx + ti] = leftTerm + rightTerm;
	}
}

/******************************************************************************
 * Step V: Launch the GPU kernel to calculate the error at each grid point    *
 *         here.                                                              *
 *****************************************************************************/
__global__ void computeJacobiError(int nx, int ny, double D, double t, double* _u, double* _error) {
	int ti = blockDim.x * blockIdx.x + threadIdx.x;
	int tj = blockDim.y * blockIdx.y + threadIdx.y;

	if (ti < (nx-1) && ti > 0 && tj < (ny-1) && tj > 0) {
		double discretizedValue = _u[tj*nx + ti];
		double analyticalValue = sin(ti)*sin(tj)*exp(-2*D*t);

		_error[tj*nx + ti] = abs(discretizedValue - analyticalValue);
	}
}

/*** Main Function ***/
int main(int argc, char *argv[])
{
  /* Variable declaration */
  double Lx = PI;           // Domain length in x-direction
  double Ly = PI;           // Domain length in y-direction
  double D = 1.;            // Diffusion constant

  int nx, ny;               // Grid points (grid cells + 1)
  double dx, dy;            // Grid spacing
  double dt;                // Time step size
  double sim_time;          // Length of sim time, arbitrary for simplicity
  double pref;              // Pre-factor in the Jacobi method

  double error = 0.;        // Mean percent-difference at each grid point
  error = error;            // To prevent compiler warning

  /* Parse command-line for problem size */
  nx = parse_cmdline(argc, argv);
  ny = nx;                  // Assume a square grid

  /* Initialize variables */
  dx = Lx / (nx - 1);       // Cell width in x-direction
  dy = Ly / (ny - 1);       // Cell width in y-direction
  dt = 0.25*dx*dy/D;        // Limited by diffusive stability
  sim_time = 0.5*Lx*Ly/D;   // Arbitrary simulation length
  pref = D*dt/(dx*dx);      // Jacobi pre-factor

  /*****************************************************************************
   * Step I: Declare, allocate, and initialize memory for the field variable   *
   *         u on the CPU.                                                     *
   ****************************************************************************/
	double* u = (double*) malloc(nx*ny * sizeof(double));
	for (int j = 0; j < ny; ++j) {
		for (int i = 0; i < nx; ++i) {
			u[j*nx + i] = sin(i) * sin(j); 
		}
	}

  /*****************************************************************************
   * Step II: Declare and allocate GPU memory for _u, _u_new, and _error. Copy *
   *          the initial condition to the GPU.                                *
   ****************************************************************************/
	double *_u, *_u_new, *_error;
	hipMalloc(&_u, nx*ny * sizeof(double));
	hipMemcpy(_u, u, nx*ny * sizeof(double), hipMemcpyHostToDevice);
	hipMalloc(&_u_new, nx*ny * sizeof(double));
	hipMalloc(&_error, nx*ny * sizeof(double));

  // Set the new soln and error to 0
  hipMemset(_u_new, 0., nx*ny * sizeof(double));
  hipMemset(_error, 0., nx*ny * sizeof(double));

  // Create thrust pointers to device memory for error calculation
  thrust::device_ptr<double> t_error(_error);

  /*****************************************************************************
   * Step III: Set up the kernel execution configuration for the domain based  *
   *           on the input domain size and the MAX_THREADS_DIM variable.      *
   ****************************************************************************/
	int tx = MAX_THREADS_DIM;
	int ty = MAX_THREADS_DIM;

	int bx = (int) ceil((double) nx / tx);
	int by = (int) ceil((double) ny / ty);

	dim3 dimBlocks(tx, ty);
	dim3 numBlocks(bx, by);

  /***************************/
  /* Main Time-Stepping Loop */
  /***************************/
  for (double time = 0.; time <= sim_time; time += dt) {
    /***************************************************************************
     * Step IV: Launch the GPU kernel to advance to the next time step with    *
     *          the Jacobi method here.                                        *
     **************************************************************************/
		computeNextJacobiStep<<<numBlocks, dimBlocks>>>(nx, ny, pref, _u, _u_new);
		hipDeviceSynchronize();

    /***************************************************************************
     * Step V: Launch the GPU kernel to calculate the error at each grid point *
     *         here.                                                           *
     **************************************************************************/
		computeJacobiError<<<numBlocks, dimBlocks>>>(nx, ny, D, time, _u, _error);
		hipDeviceSynchronize();

    // Use thrust to do a parallel reduction on the error
    error = thrust::reduce(t_error, t_error + nx*ny, 0., thrust::plus<double>());
    printf("Error at t* = %.5lf is %e\n", time*D/(Lx*Lx), error/(nx*ny));

    // Copy new soln to old. This also blocks to ensure computations are finished.
    hipMemcpy(_u, _u_new, nx*ny * sizeof(double), hipMemcpyDeviceToDevice);
  }

  /*****************************************************************************
   * Step VI: Copy the memory back to the CPU.                                 *
   ****************************************************************************/
	hipMemcpy(u, _u, nx*ny * sizeof(double), hipMemcpyDeviceToHost);

  /*****************************************************************************
   * Step I and Step II: Free the memory that you declared and allocated       *
   *                     earlier in the program.                               *
   ****************************************************************************/
	free(u);
	hipFree(_u);
	hipFree(_u_new);
	hipFree(_error);

  return EXIT_SUCCESS;
}

